#include <hip/hip_runtime.h>


/* initialize grid
 * we'll distribute all particles evenly on the screen
 */
__global__ void d_reset(float4* verts, float4* states,
                         float ww, float wh,
                         unsigned int mesh_width, unsigned int mesh_height)
{
    //grid-stride loop
    for (unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
         y < mesh_height;
         y += blockDim.y * gridDim.y)
    {
        for (unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
             x < mesh_width;
             x += blockDim.x * gridDim.x)
        {
            int vi = y*mesh_width + x;
            float u, v, w;
            u = ww * (x*1.f/mesh_width);
            v = wh * (y*1.f/mesh_height);
            w = 0.0f;
            verts[vi].x = u;
            verts[vi].y = v;
            verts[vi].z = w;
            states[vi].x = 0.0f;
            states[vi].y = 0.0f;
            states[vi].z = 0.0f;
            /* color information is uploaded by host, no init here */
        }
    }
}

/*
 * calculate particle forces & new positions & colors
 */
__global__ void d_advance(float4* verts, float4* states,
                           float mx, float my,
                           unsigned int mesh_count,
                           float speed,
                           float delta)
{
    for (unsigned int vi = blockIdx.x * blockDim.x + threadIdx.x;
         vi < mesh_count;
         vi += blockDim.x * gridDim.x)
    {
        // colors are stored after all vertices
        unsigned int ci = mesh_count + vi;

        float3 state  = {states[vi].x, states[vi].y, 0.f/*states[vi].z*/};

        // calculate vector between mouse and particle
        float3 dir_force = {verts[vi].x-mx, verts[vi].y-my, 0.f};
        /* steering:
         * the new particle state vector is given by current directional force
         * and the old state scaled by the particle mass
         */
        float flen = rsqrtf( dir_force.x*dir_force.x
                             +dir_force.y*dir_force.y
            /*+dir_force.z*dir_force.z*/);
        dir_force.x *= flen;
        dir_force.y *= flen;
        /*dir_force.z *= flen;*/
        state.x += delta * dir_force.x;
        state.y += delta * dir_force.y;
        /*state.z += dir_force.z;*/

        // speed: base speed scale by particle mass
        float v = speed / states[vi].w;
        /* update particle position:
         * new_position = old_position - state_vector*speed;
         */
        float3 dv = {state.x * v, state.y * v, 0.f/*state.z * v*/};

        verts[vi].x -= delta * dv.x;
        verts[vi].y -= delta * dv.y;
        /*verts[vi].z -= delta * dv.z;*/

        // update colors depending on particles force
        float dx = 0.5f*dv.x;
        float dy = 0.5f*dv.y;

        verts[ci].x = 0.1f+fabs(dx);
        verts[ci].y = 0.1f+fabs(dy);
        verts[ci].z = 0.1f+fabs(dx+dy);
        states[vi].x = state.x;
        states[vi].y = state.y;
        /*states[vi].z = state.z;*/
    }
}

void kernel_reset(float4* verts, float4* states,
                  int ww, int wh,
                  unsigned int mesh_width, unsigned int mesh_height,
                  unsigned int numSMs)
{
    dim3 threads(32,32);
    dim3 blocks(32*numSMs);

    d_reset<<<blocks, threads>>>( verts, states,
        ww, wh,
        mesh_width, mesh_height );
}

void kernel_advance(float4* verts, float4* states,
                    float mx, float my,
                    unsigned int mesh_count,
                    float speed,
                    unsigned int numSMs,
                    double delta)
{
    dim3 threads(1024);
    dim3 blocks( 32*numSMs );

    d_advance<<<blocks, threads>>>( verts, states,
        mx, my,
        mesh_count,
        speed,
        static_cast<float>(delta));
}